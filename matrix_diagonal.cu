#include "hip/hip_runtime.h"
/* 
By : Johan S. Suarez L. or @jadry92 in twitter
This kernel takes the diagonals elements of the matrix  A in the vector B 


*/


#include <stdio.h>              //manipulacion de ficheros, lectura-escritura ficheros, scandf-printf
#include <stdlib.h>             //Conversion de tipos de datos, memoria dinamica, abs
#include <string.h>             //Uso de memcpy principalmente
#include <math.h>				//funciones matemáticas
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// Macro for mastrix index
#define Ind(a,i,j) (a)[(j)+(i)*N]

/* Function check malloc of variables */
void checkMalloc(void * var,const char *name){
	if( var == NULL){
      	printf("It's not possible to allocate  %s. \n",name);}
    else {
     	printf(" Memory has already been allocated %s.\n",name);}
}

/* Function take the time */
double timeval_diff(struct timeval *a, struct timeval *b){
  return (double)(a->tv_sec + (double)a->tv_usec/1000000) - (double)(b->tv_sec + (double)b->tv_usec/1000000);
}

__global__ void kernel(float *d_A,float *d_B,int N){
	
	/* Thread index */
	int id = threadIdx.x + blockIdx.x*blockDim.x;

	Ind(d_B,id,id) = 1/sqrt(Ind(d_A,id,id));
}

int main(int argc, char **argv){
	/* Initiation of variables*/
	int i,j; // i == rows and j == colms
	int N =3;
	float* h_A;
    float* h_B;
	struct timeval t_ini, t_fin;
	double full_time;




    /* Reservation of memory */


    h_A = (float*)calloc(N, sizeof(float));
    	checkMalloc(h_A,"h_A");
	h_B = (float*)calloc(N, sizeof(float));
		checkMalloc(h_B,"h_B");


	/* Initialitation Matix A*/

	printf("  A =\n");
    for(i = 0; i < N; i++){
    	for(j = 0; j < N ; j++){
    		Ind(h_A,i,j) = rand()/(RAND_MAX/10.0);
    		printf(" %f ",Ind(h_A,i,j));
    	}
    printf("\n");	
    }

    memset(h_B,0.0,N*N*sizeof(float));

    /* Take Diagonal points of Matrix A and set in matrix
	B through the ecuation 1/sqrt(D(A))
    */
		/* Start take the time*/
	gettimeofday(&t_ini, NULL);

 	for(i = 0: i < N ){
 		if(Ind(h_A,i,i)!=0){
 			Ind(h_B,i,i) = 1.0/sqtr(Ind(h_A,i,i));
 		}
 	}

 	gettimeofday(&t_fin, NULL);
	full_time = timeval_diff(&t_fin, &t_ini);
	printf("Time CPU = %f [s]\n",full_time);


    printf("B =\n");
    for(i = 0; i < N; i++){
    	for(j = 0; j < N ; j++){
    		printf(" %f ",Ind(B,i,j));
    	}
    printf("\n");	
    }


	/* Free Memory*/
	free(A);
	free(B);

	return 0;
}
