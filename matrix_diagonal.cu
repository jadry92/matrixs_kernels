/* 
By : Johan S. Suarez L. or @jadry92 in twitter
This kernel takes the diagonals elements of the matrix  A in the vector B 


*/

// Macro for mastrix index
#define Ind(a,i,j) (a)[(j)+(i)*N]


#include <hip/hip_runtime.h>
#include <stdio.h>              //manipulacion de ficheros, lectura-escritura ficheros, scandf-printf
#include <stdlib.h>             //Conversion de tipos de datos, memoria dinamica, abs
#include <string.h>             //Uso de memcpy principalmente
#include <math.h>				//funciones matemáticas
#include <time.h>	

/* Funtion check malloc of variables */
void checkMalloc(void * var,const char *name){
	if( var == NULL){
      	printf("It's not possible to allocate  %s. \n",name);}
    else {
     	printf(" Memory has already been allocated %s.\n",name);}
}


int main(int argc, char **argv){
	/* Initiation of variables*/
	int i,j; // i == rows and j == colms
	int N =3;
	float* A;
    float* B;



    /* Reservation of memory */

    A = (float*)calloc(N, sizeof(float));
    	checkMalloc(A,"A");
	B = (float*)calloc(N, sizeof(float));
		checkMalloc(B,"B");

	/* Initialitation Matix A*/

	printf("A =\n");
    for(i = 0; i < N; i++){
    	for(j = 0; j < N ; j++){
    		Ind(A,i,j) = rand()/(RAND_MAX/10.0);
    		printf(" %f ",Ind(A,i,j));
    	}
    printf("\n");	
    }

    /* Gaussian Elimination*/
    // The B has to be identity matrix
    memset(B,0.0,N*sizeof(float));
    for(i = 0; i < N; i++){
    	Ind(B,i,i) = 1.0;	
    }
    printf("B =\n");
    for(i = 0; i < N; i++){
    	for(j = 0; j < N ; j++){
    		printf(" %f ",Ind(B,i,j));
    	}
    printf("\n");	
    }

    int y;
    float key;
    for(i = 0; i < N; i++){ // for to rows
    	for(j = 0; j <N  ; j++){
    		y=0;
    		if(i==j){

    		}else{
    		while(y<N){
    	 		if(i+1<N ){
    	 			Ind(A,i,j+y) = Ind(A,i,j+y) - Ind(A,i+1,j+y)*(Ind(A,i,j)/Ind(A,i+1,j));
    				Ind(B,i,j+y) = Ind(B,i,j+y) - Ind(B,i+1,j+y)*(Ind(A,i,j)/Ind(A,i+1,j));
    			}else{
    				Ind(A,i,j+y) = Ind(A,i,j+y) - Ind(A,i-N-1+y,j+y)*(Ind(A,i,j)/Ind(A,i-N-1,j));
    				Ind(B,i,j+y) = Ind(B,i,j+y) - Ind(B,i-N-1+y,j+y)*(Ind(A,i,j)/Ind(A,i-N-1,j));
    			}	
  				y++;
    		}
    		}	
    	 		//Ind(A,i,j) = 1;
    			//Ind(B,i,j) = Ind(B,i,j)/Ind(A,i,j);
    		//}
    	}
    }				

	printf("A =\n");
    for(i = 0; i < N; i++){
    	for(j = 0; j < N ; j++){
    		printf(" %f ",Ind(A,i,j));
    	}
    printf("\n");	
    }

    printf("B =\n");
    for(i = 0; i < N; i++){
    	for(j = 0; j < N ; j++){
    		printf(" %f ",Ind(B,i,j));
    	}
    printf("\n");	
    }
    /* Descompotition LU*/


	return 0;
}
