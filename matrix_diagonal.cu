#include "hip/hip_runtime.h"
/* 
By : Johan S. Suarez L. or @jadry92 in twitter
This kernel takes the diagonals elements of the matrix  A in the vector B 


*/

// Macro for mastrix index
#define Ind(a,i,j) (a)[(j)+(i)*N]

#include <stdio.h>              //manipulacion de ficheros, lectura-escritura ficheros, scandf-printf
#include <stdlib.h>             //Conversion de tipos de datos, memoria dinamica, abs
#include <string.h>             //Uso de memcpy principalmente
#include <math.h>				//funciones matemáticas
#include <time.h>	

/* Funtion check malloc of variables */
void checkMalloc(void * var,const char *name){
	if( var == NULL){
      	printf("It's not possible to allocate  %s. \n",name);}
    else {
     	printf(" Memory has already been allocated %s.\n",name);}
}


int main(int argc, char **argv){
	/* Initiation of variables*/
	int i,j; // i == rows and j == colms
	int N =3;
	float* A;
    float* B;



    /* Reservation of memory */

    A = (float*)calloc(N, sizeof(float));
    	checkMalloc(B,"B");
	B = (float*)calloc(N, sizeof(float));
		checkMalloc(A,"A");

	/* Initialitation Matix A*/

	printf("A =\n");
    for(i = 0; i < N; i++){
    	for(j = 0; j < N ; j++){
    		Ind(A,i,j) = rand()/(RAND_MAX/10.0);
    		printf(" %f ",Ind(A,i,j));
    	}
    printf("\n");	
    }

    /* Gaussian Elimination*/
    // The B has to be identity matrix

    float key;
    for(i = 0; i < N; i++){
    	for(j = 0; j < N ; j++){
    		if(i==j){
    			key = Ind(A,i,j);
    			Ind(B,i,j) = Ind(A,i,j)/key;
    		}
    			

    /* Descompotition LU*/


	return 0;
}
