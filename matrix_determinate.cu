#include "hip/hip_runtime.h"
/* 
By : Johan S. Suarez L. or @jadry92 in twitter
This kernel find the  determinte of the matrix A in variable B 


*/







#include <stdio.h>              //manipulacion de ficheros, lectura-escritura ficheros, scandf-printf
#include <stdlib.h>             //Conversion de tipos de datos, memoria dinamica, abs
#include <string.h>             //Uso de memcpy principalmente
#include <math.h>				//funciones matemáticas
#include <time.h>	
#include <hip/hip_runtime.h>       //
#include <sys/time.h>           //
#include <unistd.h>             //

// Macro for mastrix index
#define Ind(a,i,j) (a)[(j)+(i)*N]


double timeval_diff(struct timeval *a, struct timeval *b){
    return (double)(a->tv_sec + (double)a->tv_usec/1000000) - (double)(b->tv_sec + (double)b->tv_usec/1000000);
}

/* Funtion check malloc of variables */
void checkMalloc(void * var,const char *name){
	if( var == NULL){
      	printf("\nIt's not possible to allocate  %s. \n",name);}
    else {
     	printf("\nMemory has already been allocated %s.\n",name);}
}




//float det_2x2(float *A){
//    float det;
//    det = Ind(A,,)*Ind(A,,)-Ind(A,,)*Ind(A,,);
//}

int main(int argc, char **argv){
	/* Initiation of variables*/
	int i,j; // i == rows and j == colms
	int N =3;
	float* A;
    float det;
    int k;
    int count = N;
    struct timeval t_ini, t_fin;
    double time_full;   

    /* Reservation of memory */

    A = (float*)malloc(N*N*sizeof(float));
    	checkMalloc(A,"A");
    float a[9]= {0,1,2,2,5,5,7,7,0};
    	/* Initialitation Matix A*/
	memcpy(A,&a,N*N*sizeof(float));
	printf("Initialitation Matix A \n");
    for(i = 0; i < N; i++){
    	for(j = 0; j < N ; j++){
    		//Ind(A,i,j) = {0,1,2,2,5,5,7,7,0};//rand()/(RAND_MAX/10.0);
    		printf(" %f",Ind(A,i,j));
    	}
    	printf("\n");	
    }	

    printf("Strat Compute\n");
    gettimeofday(&t_ini, NULL);
    det = Ind(A,0,0);
    for (k = 0; k < count-1;++k){      
        for(i = k+1; i < N; i++){
            for(j = k+1; j < N; j++){
                Ind(A,i,j)=(Ind(A,k,k)*Ind(A,i,j)-Ind(A,k,j)*Ind(A,i,k))/Ind(A,k,k);
            }
            det = det*Ind(A,k+1,k+1);
        }    
//        det = det*Ind(A,k+1,k+1);
    }
}
    gettimeofday(&t_fin, NULL);
    time_full = timeval_diff(&t_fin, &t_ini);   
    printf("\nTime of compute =%f [s] \n",time_full);
    printf("\nTime of compute =%f [min] \n",time_full/60.0);

    printf("\ndet = %f\n",det );

}
